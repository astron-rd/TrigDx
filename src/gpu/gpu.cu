#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "gpu.cuh"

__global__ void kernel_sinf(const float *__restrict__ x, float *__restrict__ s,
                            size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    // s[idx] = __sinf(x[idx]);
    s[idx] = sinf(x[idx]);
  }
}

__global__ void kernel_cosf(const float *__restrict__ x, float *__restrict__ c,
                            size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    // c[idx] = __cosf(x[idx]);
    c[idx] = cosf(x[idx]);
  }
}

__global__ void kernel_sincosf(const float *__restrict__ x,
                               float *__restrict__ s, float *__restrict__ c,
                               size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    // __sincosf(x[idx], &s[idx], &c[idx]);
    s[idx] = sinf(x[idx]);
    c[idx] = cosf(x[idx]);
  }
}

__global__ void kernel_expf(const float *__restrict__ x, float *__restrict__ e,
                            size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    // e[idx] = __expf(x[idx]);
    e[idx] = expf(x[idx]);
  }
}

namespace {
inline dim3 make_grid(size_t n, size_t threadsPerBlock = 256) {
  return dim3((n + threadsPerBlock - 1) / threadsPerBlock);
}
} // namespace

void launch_sinf_kernel(const float *d_x, float *d_s, size_t n) {
  dim3 blocks(256);
  dim3 grid = make_grid(n, blocks.x);
  kernel_sinf<<<grid, blocks>>>(d_x, d_s, n);
}

void launch_cosf_kernel(const float *d_x, float *d_c, size_t n) {
  dim3 blocks(256);
  dim3 grid = make_grid(n, blocks.x);
  kernel_cosf<<<grid, blocks>>>(d_x, d_c, n);
}

void launch_sincosf_kernel(const float *d_x, float *d_s, float *d_c, size_t n) {
  dim3 blocks(256);
  dim3 grid = make_grid(n, blocks.x);
  kernel_sincosf<<<grid, blocks>>>(d_x, d_s, d_c, n);
}

void launch_expf_kernel(const float *d_x, float *d_e, size_t n) {
  dim3 blocks(256);
  dim3 grid = make_grid(n, blocks.x);
  kernel_expf<<<grid, blocks>>>(d_x, d_e, n);
}
